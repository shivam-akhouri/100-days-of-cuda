#include <iostream>
#include <hip/hip_runtime.h>

using namespace std;

__global__ void matrixmul(float* a, float* b, float *c, int m, int n, int k){
    int row = blockDim.y*blockIdx.y+threadIdx.y;
    int col = blockDim.x*blockIdx.x+threadIdx.x;
    if(row < m && col < n){
        float sum = 0;
        for(int l = 0; l < k; l++){
            sum += a[k*row+l]*b[l*n +col];
        }
        c[row*n+col] = sum;
    }
}

int main(){
    int m = 3, k = 2, n = 4;
    float *a, *b, *c;
    float *da, *db, *dc;
    a = (float*)malloc(sizeof(float)*m*k);
    b = (float*)malloc(sizeof(float)*k*n);
    c = (float*)malloc(sizeof(float)*m*n);
    int counter = 1;
    for(int i = 0; i< m; i++){
        for(int j = 0; j < k ;j++){
            a[i*k+j] = counter;
            counter++;
        }
    }
    counter = 7;
    for(int i = 0; i < k; i++){
        for(int j = 0; j < n; j++){
            b[i*n+j] = counter;
            counter++;
        }
    }

    hipMalloc(&da, sizeof(float)*m*k);
    hipMalloc(&db, sizeof(float)*k*n);
    hipMalloc(&dc, sizeof(float)*m*n);
    hipMemcpy(da, a, sizeof(float)*m*k, hipMemcpyHostToDevice);
    hipMemcpy(db, b, sizeof(float)*k*n, hipMemcpyHostToDevice);
    dim3 block(32, 32);
    dim3 grid((n+31)/32, (m+31)/32);
    matrixmul<<<grid, block>>>(da, db, dc, m, n, k);
    hipMemcpy(c, dc, sizeof(float)*m*n, hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    for(int i =0; i < m; i++){
        for(int j = 0; j < n; j++){
            cout << c[i*n+j] << " ";
        }
        cout << endl;
    }
    free(a);
    free(b);
    free(c);
    hipFree(da);
    hipFree(db);
    hipFree(dc);

    return 0;
}