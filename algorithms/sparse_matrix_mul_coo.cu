#include <hip/hip_runtime.h>
#include <iostream>

#define N 4  // Matrix size

// Kernel function for COO SpMV
__global__ void coo_spmv(int *rowIdx, int *colIdx, float *values, int nnz, float *x, float *y) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < nnz) {
        atomicAdd(&y[rowIdx[i]], values[i] * x[colIdx[i]]);
    }
}

int main() {
    // Example sparse matrix in COO format
    int h_rowIdx[] = {0, 0, 1, 2, 2, 3};
    int h_colIdx[] = {0, 1, 1, 2, 3, 3};
    float h_values[] = {1.0, 2.0, 3.0, 4.0, 5.0, 6.0};
    int nnz = 6;

    // Input vector
    float h_x[] = {1, 2, 3, 4};
    float h_y[N] = {0};

    // Device memory allocation
    int *d_rowIdx, *d_colIdx;
    float *d_values, *d_x, *d_y;
    
    hipMalloc(&d_rowIdx, nnz * sizeof(int));
    hipMalloc(&d_colIdx, nnz * sizeof(int));
    hipMalloc(&d_values, nnz * sizeof(float));
    hipMalloc(&d_x, N * sizeof(float));
    hipMalloc(&d_y, N * sizeof(float));

    // Copy data to device
    hipMemcpy(d_rowIdx, h_rowIdx, nnz * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_colIdx, h_colIdx, nnz * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_values, h_values, nnz * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_x, h_x, N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_y, h_y, N * sizeof(float), hipMemcpyHostToDevice);

    // Launch kernel
    coo_spmv<<<1, nnz>>>(d_rowIdx, d_colIdx, d_values, nnz, d_x, d_y);
    
    // Copy result back
    hipMemcpy(h_y, d_y, N * sizeof(float), hipMemcpyDeviceToHost);

    // Print result
    std::cout << "Result: ";
    for (int i = 0; i < N; i++)
        std::cout << h_y[i] << " ";
    std::cout << std::endl;

    // Free device memory
    hipFree(d_rowIdx);
    hipFree(d_colIdx);
    hipFree(d_values);
    hipFree(d_x);
    hipFree(d_y);

    return 0;
}