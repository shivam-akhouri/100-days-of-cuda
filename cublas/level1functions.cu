#include <iostream>
#include <hipblas.h>
#include <hip/hip_runtime.h>

using namespace std;

int main(){
    int length = 10;
    size_t size = length*sizeof(float);
    hipblasHandle_t handle;
    float *input;
    // Used for index based return values of cublas functions
    int index;
    // Used for reduction based operation values like sum, product etc.
    float result;
    input = (float*)malloc(size);
    for(unsigned int i = 0;i < length; i++){
        input[i] = rand()%100;
    }
    // Printing input array
    for(unsigned int i = 0;i < length; i++){
        cout << input[i] << " ";
    }
    cout << endl;


    float *da;
    hipMalloc(&da, size);
    hipMemcpy(da, input, size, hipMemcpyHostToDevice);
    hipblasCreate(&handle);

    hipblasIsamax(handle, length, da, 1, &index);
    cout << "Maximum in the array is: " << input[index-1] << endl;
    
    hipblasIsamin(handle, length, da, 1, &index);
    cout << "Manimum in the array is: " << input[index-1] << endl;

    hipblasSasum(handle, length, da, 1, &result);
    cout << "Sum of the array element is: " << result << endl; 

    float *dy, *output;
    output = (float*)malloc(size);
    hipMalloc(&dy, size);
    hipMemcpy(dy, input, size, hipMemcpyHostToDevice);
    float alpha = 2;

    hipblasSaxpy(handle, length, &alpha, da, 1, dy, 1);
    hipMemcpy(output, dy, size, hipMemcpyDeviceToHost);


    cout << "Result of operation alhpa*x+y: "<< endl;
    for(int i  = 0; i < length; i++){
        cout << output[i] << " ";
    }
    cout << endl;

    float c = 0.2, s = 0.9;
    // ((c, s)   --> Rotation matrix
    // (-s, c))
    hipblasStatus_t status = hipblasSrot(handle, length, da, 1, dy, 1, &c, &s);
    if(status == HIPBLAS_STATUS_SUCCESS) cout << "Success" << endl;
    else if(status == HIPBLAS_STATUS_EXECUTION_FAILED) cout << "Failed" << endl;
    
    hipMemcpy(input, da, size, hipMemcpyDeviceToHost);
    hipMemcpy(output, dy, size, hipMemcpyDeviceToHost);
    cout << "Input array 1 after rotation changes to: " << endl;
    for(int i  = 0; i < length; i++){
        cout << input[i] << " ";
    }
    cout << endl;
    cout << "Input array 2 after rotation changes to: " << endl;
    for(int i  = 0; i < length; i++){
        cout << output[i] << " ";
    }
    cout << endl;

    hipblasSscal(handle, length, &alpha, da, 1);
    hipMemcpy(input, da, size, hipMemcpyDeviceToHost);
    cout<< "Matrix after applying scaling factor: " << endl;
    for(int i  = 0; i < length; i++){
        cout << input[i] << " ";
    }
    cout << endl;
    cout << "Element is vector 1 before swapping" << endl;
    for(unsigned int i = 0; i < length; i++){
        cout << input[i] << " ";
    }
    cout << endl;
    cout << "Element is vector 2 before swapping" << endl;
    for(unsigned int i = 0; i < length; i++){
        cout << output[i] << " ";
    }
    cout << endl;

    hipMemcpy(da, input, size, hipMemcpyHostToDevice);
    hipMemcpy(dy, output, size, hipMemcpyHostToDevice);
    hipblasSswap(handle, length, da, 1, dy, 1);
    hipMemcpy(input, da, size, hipMemcpyDeviceToHost);
    hipMemcpy(output, dy, size, hipMemcpyDeviceToHost);

    cout << "Element is vector 1 after swapping" << endl;
    for(unsigned int i = 0; i < length; i++){
        cout << input[i] << " ";
    }
    cout << endl;
    cout << "Element is vector 2 after swapping" << endl;
    for(unsigned int i = 0; i < length; i++){
        cout << output[i] << " ";
    }
    cout << endl;
    
    hipblasDestroy(handle);
    hipFree(da);
    free(input);

    return 0;
}