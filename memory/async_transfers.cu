#include <iostream>
#include <hip/hip_runtime.h>

#define BLOCK_SIZE 128

using namespace std;

void init_vector(float* arr, int length, bool flag=false){
    for(int i = 0;i < length; i++){
        arr[i] = i;
        if(flag) arr[i] = 0;
    }
}

__global__ void vectoradd(float *a, float *b, float *c, int length){
    int idx = blockDim.x*blockIdx.x + threadIdx.x;
    if(idx < length) c[idx] = a[idx]+b[idx];
}

int main(){
    int length = 100000;
    float *a, *b, *c;
    float *da, *db, *dc;

    a = (float*)malloc(sizeof(float)*length);
    b = (float*)malloc(sizeof(float)*length);
    c = (float*)malloc(sizeof(float)*length);
    hipMalloc(&da, sizeof(float)*length);
    hipMalloc(&db, sizeof(float)*length);
    hipMalloc(&dc, sizeof(float)*length);

    init_vector(a, length);    
    init_vector(b, length);    
    init_vector(c, length, true);
    
    hipStream_t stream1, stream2;
    hipStreamCreate(&stream1);
    hipStreamCreate(&stream2);

    hipEvent_t start, end;
    hipEventCreate(&start);
    hipEventCreate(&end);


    // Both A and B will be transferred at the same time and not in sequential order.
    hipEventRecord(start);
    hipMemcpyAsync(da, a, sizeof(float)*length, hipMemcpyHostToDevice, stream1);
    hipMemcpyAsync(db, b, sizeof(float)*length, hipMemcpyHostToDevice, stream2);
    hipEventRecord(end);

    hipStreamSynchronize(stream1);
    hipStreamSynchronize(stream2);

    dim3 block(BLOCK_SIZE, 1,1);
    dim3 grid((length+BLOCK_SIZE-1)/BLOCK_SIZE, 1, 1);
    vectoradd<<<grid, block>>>(da, db, dc, length);

    hipMemcpyAsync(c, dc, sizeof(float)*length, hipMemcpyDeviceToHost, stream1);
    hipStreamSynchronize(stream1);
    
    float duration;
    hipEventElapsedTime(&duration, start, end);
    cout << "Copy command took: " << duration << " milliseconds" << endl;
    // for(int i =0; i < length; i++){
    //     cout << c[i] << " ";
    // }
    return 0;
}